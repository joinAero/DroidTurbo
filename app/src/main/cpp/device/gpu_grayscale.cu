#include "hip/hip_runtime.h"
#include "gpu_grayscale.h"
#include "gpu_helper.h"

__global__ void kernel_grayscale(
        rgba_t * const rgba_pixels, const uint32_t n) {
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    //const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    //const unsigned int tid = ((gridDim.x * blockDim.x) * idy) + idx;
    if (idx < n) {
        // Y = 0.299*R + 0.587*G + 0.114*B
        rgba_t &rgba = rgba_pixels[idx];
        uint8_t gray = (uint8_t) (0.299 * rgba.r + 0.587 * rgba.g + 0.114 * rgba.b);
        rgba.r = rgba.g = rgba.b = gray;
    }
}

void gpu_grayscale(const AndroidBitmapInfo &info, rgba_t *rgba_pixels) {
    const uint32_t n = info.width * info.height;
    GpuArray<rgba_t> gpu_rgba_pixels(n);
    gpu_rgba_pixels.Set(rgba_pixels, n);
    TIME_BEG("kernel_grayscale");
    kernel_grayscale<<<n/256+1, 256>>>(gpu_rgba_pixels.GetData(), n);
    TIME_END("kernel_grayscale");
    gpu_rgba_pixels.Get(rgba_pixels, n);
}
