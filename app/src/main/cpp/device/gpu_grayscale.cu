#include "hip/hip_runtime.h"
#include "gpu_grayscale.h"
#include "gpu.h"

__global__ void kernel_grayscale(
        uint32_t * const rgba_pixels, const uint32_t n) {
    const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const unsigned int tid = ((gridDim.x * blockDim.x) * idy) + idx;
    if (tid < n) {
        // Y = 0.299*R + 0.587*G + 0.114*B
        uint32_t &rgba = rgba_pixels[tid];
        uint32_t gray = (uint32_t) (
                0.299 * ((rgba >> 24) & 0xFF) +
                0.587 * ((rgba >> 16) & 0xFF) +
                0.114 * ((rgba >> 8) & 0xFF));
        rgba = ((gray & 0xFF) << 24) +
                ((gray & 0xFF) << 16) +
                ((gray & 0xFF) << 8) +
                (gray & 0xFF);
    }
}

void gpu_grayscale(const AndroidBitmapInfo &info, uint32_t *rgba_pixels) {
     const uint32_t n = info.width * info.height;
     GpuArray<uint32_t> gpu_rgba_pixels(n);
     gpu_rgba_pixels.Set(rgba_pixels, n);
     kernel_grayscale<<<n/256+1, 256>>>(gpu_rgba_pixels.GetData(), n);
     gpu_rgba_pixels.Get(rgba_pixels, n);
}
